#include "hip/hip_runtime.h"
#include "ppmFile.h"
#include "ppmFile.c"
#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

Image *mean(Image *imgin, int r);

__global__ void row(int n, int r, int width, int height, unsigned char *in, unsigned char *out) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        int y = i / width;
        int x = i % width;
        int left = (x - r) * (r < x);
        int right = (width - 1) * (x + r >= width - 1) + (x + r) * (x + r < width - 1);

        int count = (right - left) + 1;

        int sumr = 0;
        int sumg = 0;
        int sumb = 0;

        for (int cx = left; cx <= right; cx++) {
            int ci = y * width + cx;
            sumr += in[ci*3];
            sumg += in[ci*3+1];
            sumb += in[ci*3+2];
        }
        out[(y * width + x)*3] = sumr / count;
        out[(y * width + x)*3 + 1] = sumg / count;
        out[(y * width + x)*3 + 2] = sumb / count;
    }
  }


__global__ void col(int n, int r, int width, int height, unsigned char *in, unsigned char *out) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        int y = i / width;
        int x = i % width;
        int top = (y - r) * (r < y);
        int bottom = (height - 1) * (y + r >= height - 1) + (y + r) * (y + r < height - 1);

        int count = (bottom - top) + 1;

        int sumr = 0;
        int sumg = 0;
        int sumb = 0;

        for (int cy = top; cy <= bottom; cy++) {
            int ci = cy * width + x;
            sumr += in[ci*3];
            sumg += in[ci*3+1];
            sumb += in[ci*3+2];
        }

        out[(y * width + x)*3] = sumr / count;
        out[(y * width + x)*3 + 1] = sumg / count;
        out[(y * width + x)*3 + 2] = sumb / count;
    }
}

/*int main(int argc, char *argv[]) {
  int r = atoi(argv[1]);
  Image *unfilterImage = ImageRead(argv[2]);
  Image *newIm;
  int imageH = ImageHeightHost(unfilterImage);
  int imageW = ImageWidthHost(unfilterImage);
  newIm = filter(imageW, imageH, r, unfilterImage);
  ImageWrite(newIm, argv[3]);
}*/

int main(int argc, char *argv[]) {
   
    

    const char *input = argv[2];
    const char *output = argv[3];
    int r = atoi(argv[1]);

    
    Image *in = ImageRead(input);

    Image *out = mean(in, r);
    ImageWrite(out, output);

    return 0;
}

Image *mean(Image *imgin, int r) {
    int width = imgin->width;
    int height = imgin->height;
    int size = width * height;
    
    unsigned char *imagein = imgin->data;

    Image *imgout = ImageCreate(width, height);
    unsigned char *imageout = imgout->data;

    unsigned char *gpuin;
    unsigned char *gpuint;
    unsigned char *gpuout;

    hipMalloc((void **)&gpuin, size * 3* sizeof(unsigned char));
    hipMalloc((void **)&gpuint, size * 3 *sizeof(unsigned char));
    hipMalloc((void **)&gpuout, size * 3 *sizeof(unsigned char));

    hipMemcpy(gpuin, imagein, size *3* sizeof(unsigned char), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;

    row<<<numBlocks, blockSize>>>(size, r, width, height, gpuin, gpuint);
    hipDeviceSynchronize();
    col<<<numBlocks, blockSize>>>(size, r, width, height, gpuint, gpuout);
    hipDeviceSynchronize();

    hipMemcpy(imageout, gpuout, size *3* sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(gpuin);
    hipFree(gpuint);
    hipFree(gpuout);

    return imgout;
}


